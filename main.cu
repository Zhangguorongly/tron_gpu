#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <chrono>
#include <hip/hip_runtime.h>
#include "tron.h"

// GPU 内核 - 每个线程生成一个私钥并检查地址
__global__ void search_tron_vanity(uint64_t start_nonce, int target_len, int *found, char *found_priv, char *found_addr) {
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x + start_nonce;
    if (*found) return;

    uint8_t priv[32];
    gen_private_key(idx, priv);

    char addr[40];
    tron_address_from_priv(priv, addr);

    // 检查尾部是否全相等
    int len = strlen(addr);
    bool match = true;
    char last = addr[len - 1];
    for (int i = len - target_len; i < len; i++) {
        if (addr[i] != last) { match = false; break; }
    }

    if (match) {
        if (atomicCAS(found, 0, 1) == 0) {
            priv_to_hex(priv, found_priv);
            strcpy(found_addr, addr);
        }
    }
}

int main() {
    int N;
    std::cout << "输入尾部连续位数 N: ";
    std::cin >> N;

    int threads_per_block = 256;
    int blocks = 1024; // 每批生成 256 * 1024 ≈ 26 万个私钥

    int *d_found;
    char *d_priv, *d_addr;
    hipMallocManaged(&d_found, sizeof(int));
    hipMallocManaged(&d_priv, 65);
    hipMallocManaged(&d_addr, 50);
    *d_found = 0;

    uint64_t nonce = 0;
    auto start = std::chrono::high_resolution_clock::now();

    while (!*d_found) {
        search_tron_vanity<<<blocks, threads_per_block>>>(nonce, N, d_found, d_priv, d_addr);
        hipDeviceSynchronize();
        nonce += (uint64_t)threads_per_block * blocks;
    }

    auto end = std::chrono::high_resolution_clock::now();
    double secs = std::chrono::duration<double>(end - start).count();

    std::cout << "\n=== 找到匹配地址 ===\n";
    std::cout << "私钥(hex): " << d_priv << "\n";
    std::cout << "TRON 地址: " << d_addr << "\n";
    std::cout << "用时: " << secs << " 秒\n";

    FILE *fp = fopen("found.txt", "w");
    if (fp) {
        fprintf(fp, "Private Key: %s\nAddress: %s\n", d_priv, d_addr);
        fclose(fp);
    }

    hipFree(d_found);
    hipFree(d_priv);
    hipFree(d_addr);
    return 0;
}
